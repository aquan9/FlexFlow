#include "hip/hip_runtime.h"
/* Copyright 2023 CMU, Facebook, LANL, MIT, NVIDIA, and Stanford (alphabetical)
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flexflow/ffconst_utils.h"
#include "flexflow/ops/argmax.h"
#include "flexflow/utils/cuda_helper.h"

namespace FlexFlow {

__global__ void
    half_2_float_array(half *ptr, float *ptr_f, int num_of_elements) {
  CUDA_KERNEL_LOOP(i, num_of_elements) {
    ptr_f[i] = __half2float(ptr[i]);
  }
}

/*static*/
template <typename DT>
void ArgMax::forward_kernel(ArgMaxMeta const *m,
                            DT *input_ptr,
                            int *indices_ptr,
                            DT *prob_ptr,
                            int *parent,
                            int const length,
                            int const batch_size,
                            hipStream_t stream) {

  checkCUDNN(hipdnnSetStream(m->handle.dnn, stream));
  DT alpha = 1.0f, beta = 0.0f;
  if (m->beam_search) {
    // set all parents id zero in arg top1 case.
    checkCUDA(hipMemset(parent, 0, batch_size * sizeof(int)));
  }
  checkCUDNN(hipdnnReduceTensor(m->handle.dnn,
                               m->reduceMaxDesc,
                               indices_ptr /*indices*/,
                               batch_size * sizeof(int) /*indicesSizeInBytes*/,
                               m->handle.workSpace,
                               m->handle.workSpaceSize,
                               &alpha,
                               m->inputTensor,
                               input_ptr,
                               &beta,
                               m->outputTensor,
                               prob_ptr));
}

/*static*/
void ArgMax::forward_kernel_wrapper(ArgMaxMeta const *m,
                                    GenericTensorAccessorW const &input,
                                    GenericTensorAccessorW const &indices,
                                    GenericTensorAccessorW const &value,
                                    GenericTensorAccessorW const &parent) {
  hipStream_t stream;
  checkCUDA(get_legion_stream(&stream));

  hipEvent_t t_start, t_end;
  if (m->profiling) {
    hipEventCreate(&t_start);
    hipEventCreate(&t_end);
    hipEventRecord(t_start, stream);
  }
  int length = input.domain.hi()[0] - input.domain.lo()[0] + 1;
  int batch_size = input.domain.get_volume() / length;

  if (input.data_type == DT_HALF) {
    ArgMax::forward_kernel<half>(m,
                                 input.get_half_ptr(),
                                 indices.get_int32_ptr(),
                                 value.get_half_ptr(),
                                 m->beam_search ? parent.get_int32_ptr()
                                                : nullptr,
                                 length,
                                 batch_size,
                                 stream);
    if (m->beam_search) {
      half_2_float_array<<<GET_BLOCKS(batch_size),
                           CUDA_NUM_THREADS,
                           0,
                           stream>>>(
          value.get_half_ptr(), m->probs, batch_size);
    }

  } else if (input.data_type == DT_FLOAT) {
    ArgMax::forward_kernel<float>(m,
                                  input.get_float_ptr(),
                                  indices.get_int32_ptr(),
                                  value.get_float_ptr(),
                                  m->beam_search ? parent.get_int32_ptr()
                                                 : nullptr,
                                  length,
                                  batch_size,
                                  stream);
  } else {
    assert(false && "Unsupported data type");
  }

  if (m->profiling) {
    hipEventRecord(t_end, stream);
    checkCUDA(hipEventSynchronize(t_end));
    float elapsed = 0;
    checkCUDA(hipEventElapsedTime(&elapsed, t_start, t_end));
    hipEventDestroy(t_start);
    hipEventDestroy(t_end);
    printf("[ArgMax] forward time = %.2lfms\n", elapsed);
  }
}

ArgMaxMeta::ArgMaxMeta(FFHandler handler,
                       Op const *op,
                       Legion::Domain const &input_domain,
                       Legion::Domain const &output_domain,
                       GenericTensorAccessorW input)
    : OpMeta(handler, op) {
  DataType data_type = op->data_type;
  checkCUDNN(hipdnnCreateTensorDescriptor(&inputTensor));
  checkCUDNN(hipdnnCreateTensorDescriptor(&outputTensor));
  checkCUDNN(hipdnnCreateReduceTensorDescriptor(&reduceMaxDesc));

  // Float and Half use save type, according to
  // https://docs.nvidia.com/deeplearning/cudnn/api/index.html#hipdnnReduceTensor:~:text=not%20coordinate%20tuples.-,The%20data%20types%20of%20the%20tensors,.,-Note%3A
  hipdnnDataType_t cudnn_data_type = HIPDNN_DATA_FLOAT;

  checkCUDNN(
      hipdnnSetReduceTensorDescriptor(reduceMaxDesc,
                                     HIPDNN_REDUCE_TENSOR_MAX,
                                     cudnn_data_type,
                                     HIPDNN_PROPAGATE_NAN,
                                     HIPDNN_REDUCE_TENSOR_FLATTENED_INDICES,
                                     HIPDNN_32BIT_INDICES));
  checkCUDNN(cudnnSetTensorDescriptorFromDomain(
      outputTensor, output_domain, data_type));
  checkCUDNN(
      cudnnSetTensorDescriptorFromDomain(inputTensor, input_domain, data_type));

  checkCUDA(hipMalloc(&probs, sizeof(float) * BatchConfig::MAX_NUM_TOKENS));
}

}; // namespace FlexFlow